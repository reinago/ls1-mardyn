#include <hip/hip_runtime.h>
#include <stdio.h>

class Mutex {
public:
	volatile uint lockValue;

public:
	__device__ void init() {
		lockValue = 0u;
	}

	__device__ void lock() {
		while( atomicExch( (uint*) &lockValue, 1u ) != 0u )
			;
	}

	__device__ void unlock() {
		lockValue = 0u;
	}
};

static __device__ Mutex globalMutex;
static volatile __device__ uint globalCounter;

#define WARP_PRINTF( fmt, ... ) printf( "(%i %i) " fmt, blockIdx.x, threadIdx.y, ##__VA_ARGS__ )

static __global__ void init() {
	globalMutex.lockValue = 0;
	globalCounter = 0;
}

static __global__ void printResults() {
	printf( "%i\n", globalCounter );
}

static __global__ void kernel() {
	if( threadIdx.x == 0 ) {
		globalMutex.lock();

		//WARP_PRINTF( "got lock\n" );
		globalCounter++;

		//WARP_PRINTF( "incremented globalCounter\n" );
		__threadfence();

		//WARP_PRINTF( "releasing lock\n" );
		globalMutex.unlock();
	}
}

void testSingleLock() {
	init<<<1,1>>>();
	dim3 blockDim = dim3( 32, 16, 1 );
	kernel<<<16, blockDim>>>();
	printResults<<<1,1>>>();
}

